#include <iostream>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/device_ptr.h>
#include <thrust/remove.h>
#include <thrust/tuple.h>
#include <thrust/sequence.h>

typedef thrust::tuple<int, int, int, bool> XYZFlag;

struct should_remove {
  __host__ __device__
  bool operator() (const XYZFlag& tup) {
    const bool flag = thrust::get<3>(tup);
    return !flag;
  }
};

int main() {
  const int N = 4000000;
  int* x_raw_ptr;
  hipMalloc(&x_raw_ptr, N * sizeof(int));
  int* y_raw_ptr;
  hipMalloc(&y_raw_ptr, N * sizeof(int));
  int* z_raw_ptr;
  hipMalloc(&z_raw_ptr, N * sizeof(int));
  bool* should_keep_raw;
  hipMalloc(&should_keep_raw, N * sizeof(bool));

  // bind device_ptr
  thrust::device_ptr<int> x_dev_ptr(x_raw_ptr);
  thrust::device_ptr<int> y_dev_ptr(y_raw_ptr);
  thrust::device_ptr<int> z_dev_ptr(z_raw_ptr);
  thrust::device_ptr<bool> should_keep_dev_ptr(should_keep_raw); 

  // init value in dev_ptr s for test
  thrust::sequence(x_dev_ptr, x_dev_ptr + N);
  thrust::sequence(y_dev_ptr, y_dev_ptr + N, N);
  thrust::sequence(z_dev_ptr, z_dev_ptr + N, N*2);
  thrust::fill(should_keep_dev_ptr, should_keep_dev_ptr+N, false);

  //for (int i = 0; i < N; i++) {
  for (int i = 0; i < 10240; i++) {
    //x_dev_ptr[i] = i;
    //y_dev_ptr[i] = i + N;
    //z_dev_ptr[i] = i + N * 2;
    should_keep_dev_ptr[i] = (bool)(i % 2);
  }

  // remove if

  auto first = thrust::make_zip_iterator(thrust::make_tuple(x_dev_ptr, y_dev_ptr, z_dev_ptr, should_keep_dev_ptr));
  auto last = thrust::make_zip_iterator(thrust::make_tuple(x_dev_ptr+N, y_dev_ptr+N,z_dev_ptr+N, should_keep_dev_ptr+N));

  auto newEnd = thrust::remove_if(first, last, should_remove());

  // print result

  for (int i = 0; i < 2; i++) {
    std::cout << "x: " << x_dev_ptr[i];
    std::cout << ", y: " << y_dev_ptr[i];
    std::cout << ", z: " << z_dev_ptr[i];
    std::cout << ", flag: " << should_keep_dev_ptr[i] << std::endl;
  }


  return 0;
}